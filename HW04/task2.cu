#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "stencil.cuh"
// #define check

void init (float *image, float *mask, int n, int R) {
	int _2R_1 = 2*R+1;

    for (int i = 0; i < n; i++){
		image[i] = (((float)rand()/RAND_MAX)*2)-1; // range [-1, 1];
	}
	for (int i = 0; i < _2R_1; i++){
		mask[i] = (((float)rand()/RAND_MAX)*2)-1; // range [-1, 1];
	}
}

void init_check (float *image, float *mask, int n, int R) {
	int _2R_1 = 2*R+1;

    for (int i = 0; i < n; i++){
		image[i] = 2;
	}
	for (int i = 0; i < _2R_1; i++){
		mask[i] = 1;
	}
}


int main(int argc, char* argv[]){
	int n = atoi(argv[1]);
    int R = atoi(argv[2]);
    unsigned int  threads_per_block = atoi(argv[3]);
    int _2R_1 = 2*R+1;

    float *image  = (float*)malloc(n*sizeof(float)); 
    float *mask   = (float*)malloc((_2R_1)*sizeof(float)); 
    float *output = (float*)malloc(n*sizeof(float)); memset(output, 0, n*sizeof(float));
    float *image_gpu;   hipMalloc((void**)&image_gpu, n*sizeof(float));
    float *mask_gpu;   hipMalloc((void**)&mask_gpu, (_2R_1)*sizeof(float));
    float *output_gpu; hipMalloc((void**)&output_gpu, n*sizeof(float));
    
    srand(time(NULL));
#ifdef check
    init_check(image, mask, n, R);
#else 
    init(image, mask, n, R);
#endif
    

    hipMemcpy(image_gpu,  image,  n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mask_gpu,   mask,   _2R_1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(output_gpu, output, n*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipEvent_t start;
    hipEvent_t stop;
    float ms; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
        
        stencil(image_gpu, mask_gpu, output_gpu, n, R, threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(output, output_gpu, n*sizeof(float), hipMemcpyDeviceToHost);


    // TODO: change to .2
#ifdef check 
    printf("%lf\n", output[n-1]); // Print the last element of the resulting matrix.
    printf("%lf\n", ms); // Print the time taken to execute your matmul function in milliseconds using CUDA events.
#else 
    printf("%.2lf\n", output[n-1]); // Print the last element of the resulting matrix.
    printf("%.2lf\n", ms); // Print the time taken to execute your matmul function in milliseconds using CUDA events.
#endif 

    free (image);
    free (mask);
    free (output);
    hipFree(image_gpu);
    hipFree(mask_gpu);
    hipFree(output_gpu);

    return 0;
}