#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {
    
    /* format: hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, lda, B, ldb, beta, C, ldc);
    HIPBLAS_OP_N: non-transport, 
    lda: leading dimension of A, B, C
    C = (alpha)*A*B + (beta)*C
    */
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1, A, n, B, n, 0, C, n);
    float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);
}