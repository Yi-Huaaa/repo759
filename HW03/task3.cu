#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "vscale.cuh"

// #define check
// #define exp2


void init (float *a, float *b, int n) {
	for (int i = 0; i < n; i++){
        a[i] = (((float)rand()/RAND_MAX)*20)-10; // range [-10.0, 10.0]
        b[i] = (((float)rand()/RAND_MAX)); // range [0.0, 1.0]
    }
}

int main(int argc, char* argv[]){
	int n = atoi(argv[1]);

    float *a = (float*)malloc(n*sizeof(float));
    float *b = (float*)malloc(n*sizeof(float));
    float *da; hipMalloc((void**)&da, n*sizeof(float));
    float *db; hipMalloc((void**)&db, n*sizeof(float));
    
    srand(time(NULL));
    init (a, b, n);
    hipMemcpy(da, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n*sizeof(float), hipMemcpyHostToDevice);

    int num_thd, num_blk;
    num_thd = 512, num_blk = int(n/num_thd);
#ifdef check
    printf("n = %d #num_thd = %d, num_blk = %d\n", n, num_thd, num_blk);
    printf("a = [");
    for (auto i = 0; i < n; i++) {
        printf("%3lf, ", a[i]);
    } printf("]\nb = [");
    for (auto i = 0; i < n; i++) {
        printf("%3lf, ", b[i]);
    } printf("]\n");
#endif

    hipEvent_t start;
    hipEvent_t stop;
    float ms; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
        
        vscale <<< num_blk, num_thd >>> (da, db, n);
        
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipDeviceSynchronize();
    hipMemcpy(b, db, n*sizeof(float), hipMemcpyDeviceToHost);

    // print
    printf("%3lf\n", ms);
    printf("%1lf\n", b[0]);
    printf("%1lf\n", b[n-1]);

#ifdef check
    printf("output_b = [");
    for (auto i = 0; i < n; i++) {
        printf("%3lf, ", b[i]);
    } printf("]\n");
#endif
    
#ifdef exp2
    // experiment-2 
    hipMemcpy(db, b, n*sizeof(float), hipMemcpyHostToDevice);
    num_thd = 16, num_blk = int(n/num_thd);
    printf("16 threads, num_thd =%d, num_blk = %d\n", num_thd, num_blk);
  
    ms = 0.0;
    hipEventRecord(start);
    
        vscale <<< num_blk, num_thd >>> (da, db, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipDeviceSynchronize();
    printf("%3lf\n", ms);

#endif 
    
    free (a);
    free (b);
    hipFree(da);
    hipFree(db);

	return 0;
}
