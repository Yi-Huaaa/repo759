#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// #define check

__global__ void kernel (int a, int *dA) {
    int x = threadIdx.x, y = blockIdx.x, idx = threadIdx.x + blockDim.x*blockIdx.x;
    dA[idx] = a*x+y;
}

int main(int argc, char* argv[]){

    int *hA = (int*)malloc(16*sizeof(int)); memset(hA, 0, 16*sizeof(int));
    int *dA; hipMalloc((void**)&dA, 16*sizeof(int));
    hipMemcpy(dA, hA, 16*sizeof(int), hipMemcpyHostToDevice);

    srand(time(NULL));
    int a = rand() % ((1<<25)-1);
#ifdef check
    printf("a = %d\n", a);
#endif

    kernel <<< 2, 8 >>> (a, dA);

    hipDeviceSynchronize();
    hipMemcpy(hA, dA, 16*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < 16; i++) 
        printf("%d ", hA[i]);

    free (hA);
    hipFree (dA);

	return 0;
}
